#include "hip/hip_runtime.h"

//#include <stdlib.h>
#include <stdio.h>
//#include <math.h>
//#include <time.h>
//#include <cstring>
#include "locale.h"
#include <malloc.h>
using namespace std;

#define GridNx 5 // ����������� ��������� ����� �� ��� x
#define GridNy 6 // ����������� ��������� ����� �� ��� y
#define GridNz 10 // ����������� ��������� ����� �� ��� z
#define GridN GridNx*GridNy*GridNz // ��������� ����� ����� ��������� �����
#define GridXY GridNx * GridNy // ����� ����� � ��������� XY, �.�. � ����� ���� �� Z

#define CudaCoresNumber 192 // ���������� ���� cuda (https://geforce-gtx.com/710.html - ��� GT710, ��� ������ ���������� ���������� ��������)


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

int Add2Vectors(bool& retflag);
void Print3dArray(int* host_c);
void ConveyorTest();


/// <summary>
/// ���������� ��������� �������������
/// </summary>
void ShowVideoadapterProperties() {
    hipDeviceProp_t prop;
    int count;
    hipGetDeviceCount(&count);
    for (size_t i = 0; i < count; i++)
    {
        hipGetDeviceProperties(&prop, i);
        printf("������������ ����������:        %s\n", prop.name);
        printf("�������������� �����������:     %d.%d\n", prop.major, prop.minor);
        printf("���������� �����������������:   %d\n", prop.multiProcessorCount);
        printf("������ warp'�:                  %d\n", prop.warpSize);
    }

    printf("���������� ���� cuda: %d (�������� �� ������������ � �������������)\n", CudaCoresNumber);
}

/// <summary>
/// ���������� ��������� �����
/// </summary>
void ShowGridProperties()
{
    printf("\n--------------�������������� ��������� �����----------------\n");
    printf("����������� ��������� ����� �� ��� x:                %d\n", GridNx);
    printf("����������� ��������� ����� �� ��� y:                %d\n", GridNy);
    printf("����������� ��������� ����� �� ��� z:                %d\n", GridNz);
    printf("��������� ����� ����� ��������� �����:               %d\n", GridN);
    printf("����� ����� � ��������� XY, �.�. � ����� ���� �� Z:  %d\n", GridXY);
    printf("----------------------------------------------------------\n");
}

int main()
{
    // ��������� ��������� ��������� � �������
    setlocale(LC_CTYPE, "rus");
    // ����������� ���������� ����������
    ShowVideoadapterProperties();
    // ����������� ���������� �����
    ShowGridProperties();     
    // ���� ��������� ����������
    ConveyorTest();

    bool retflag;
    int retval = Add2Vectors(retflag);
    if (retflag) return retval;

    return 0;
}

/// <summary>
/// ������������� ������� �� GPU
/// </summary>
/// <param name="c"></param>
/// <param name="size"></param>
/// <returns></returns>
__global__ void initVectorInGpuKernel(int* c, unsigned int size)
{
    // Compute the offset in each dimension
    const size_t offsetX = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t offsetY = blockDim.y * blockIdx.y + threadIdx.y;
    const size_t offsetZ = blockDim.z * blockIdx.z + threadIdx.z;

    // Make sure that you are not actually outs
    if (offsetX >= GridNx || offsetY >= GridNy || offsetZ >= GridNz)
        return;

    // Compute the linear index assuming that X,Y then Z memory ordering
    const size_t idx = offsetZ * GridNx * GridNy + offsetY * GridNx + offsetX;
    printf("blockIdx.x = %d, blockIdx.y = %d, i = %d\n", blockIdx.x, blockIdx.y, idx);
    printf("offsetX = %d, offsetY = %d, offsetZ = %d \n", offsetX, offsetY, offsetZ);
    
    long nodeIndex = idx;
    for (size_t z = 0; z < GridNz; z++)
    {
        nodeIndex += GridNx * GridNy;
        if (idx < size)
        {
            c[nodeIndex] = nodeIndex;
        }
    }        
}

/// <summary>
/// ����������� � 2 ���� �������� � ���������, ����� �������� ������� ����� s
/// </summary>
/// <param name="c"></param>
/// <param name="size">���-�� ��������� ������� c</param>
/// <param name="s">i+j+k</param>
/// <returns></returns>
__global__ void conveyorKernel(int* c, unsigned int size, int s)
{
    // Compute the offset in each dimension
    const size_t offsetX = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t offsetY = blockDim.y * blockIdx.y + threadIdx.y;
    const size_t offsetZ = blockDim.z * blockIdx.z + threadIdx.z;

    // Make sure that you are not actually outs
    if (offsetX >= GridNx || offsetY >= GridNy || offsetZ >= GridNz)
        return;

    // Compute the linear index assuming that X,Y then Z memory ordering
    const size_t idx = offsetZ * GridNx * GridNy + offsetY * GridNx + offsetX;
    //printf("blockIdx.x = %d, blockIdx.y = %d, i = %d\n", blockIdx.x, blockIdx.y, idx);
    //printf("offsetX = %d, offsetY = %d, offsetZ = %d \n", offsetX, offsetY, offsetZ);

    long nodeIndex = idx;    
    for (size_t z = 0; z < GridNz; z++)
    {
        nodeIndex += GridNx * GridNy;
        if (idx < size && (offsetX + offsetY + offsetZ) == s)
        {
            c[nodeIndex] = c[nodeIndex] * 2;
        }
    }
}

void Print3dArray(int* host_c)
{
    for (size_t k = 0; k < GridNz; k++)
    {
        printf("\n--------------------------------------\n");
        printf("------------ k = %d ------------------\n", k);
        printf("--------------------------------------\n");
        for (size_t j = 0; j < GridNy; j++)
        {
            printf("------------ j = %d ------------------\n", j);
            for (size_t i = 0; i < GridNx; i++)
            {
                printf("%d\t", host_c[i + j * GridNx + k * GridXY]);
            }
            printf("\n");
        }
    }
}

void ConveyorTest()
{
    printf("------------------���� ������������ ����������----------------\n");
    // 1. ��������� ������ ������� ������
    int size = GridN; // ���-�� ���������
    size_t sizeInBytesInt = size * sizeof(int);// ������ ������� � ������

    // 2. �������� ������ ��� ������ � ���
    int* host_c = 0;
    host_c = (int*)malloc(sizeInBytesInt);

    // 3. �������� ������ ��� ������ �� ����������        
    int* dev_c = 0;
    hipError_t cudaStatus = hipMalloc((void**)&dev_c, sizeInBytesInt);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "!!!!!!!!!!!!hipMalloc failed in ConveyorTest()!!!!!!!!!!!!");
        return;
    }
    
    // 3. �������������� ������ �� GPU
    initVectorInGpuKernel <<< dim3(GridNx, GridNy), 1 >>> (dev_c, GridN);
    hipDeviceSynchronize();

    // 4. ����� ���������
    int s = 1;
    //conveyorKernel <<< dim3(GridNx, GridNy), 1 >>> (dev_c, GridN, s);
    hipDeviceSynchronize();

    hipMemcpy(host_c, dev_c, sizeInBytesInt, hipMemcpyDeviceToHost);
        
    Print3dArray(host_c);
    

    // ������� ������ ������
    free(host_c);
    hipFree(dev_c);

    printf("--------------���� ������������ ���������� (�����)------------\n");
}



__global__ void conveyorTestKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}




/// <summary>
/// �������� ���� ��������
/// </summary>
/// <param name="retflag"></param>
/// <returns></returns>
int Add2Vectors(bool& retflag)
{
    retflag = true;
    const int arraySize = GridN;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    retflag = false;
    return {};
}

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<size, 1>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
