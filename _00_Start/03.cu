
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void cuda_threadIdx(float* A, float* B, float* C, int size){    
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main() {
    // Размерность массива
    int N = 10;

    // Выделение памяти в ОЗУ
    float *a = (float*)malloc(N*sizeof(float));
    float *b = (float*)malloc(N*sizeof(float));
    float *c = (float*)malloc(N*sizeof(float));

    // Выделение памяти в GPU
    float *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, N*sizeof(float));
    hipMalloc((void**)&dev_b, N*sizeof(float));
    hipMalloc((void**)&dev_c, N*sizeof(float));

    printf("RAM massives initialization\n");
    printf("a\tb\tc\n");
    for(int i=0;i<N;i++)
    {
        a[i] = i;
        b[i] = 0.2*i;
        c[i] = 0;
        printf("%g\t%g\t%g\n", a[i], b[i], c[i]);
    }
    
    hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N*sizeof(float), hipMemcpyHostToDevice);

    int numBlocks = 1;
    dim3 threadsPerBlock(N);

    cuda_threadIdx<<<numBlocks,threadsPerBlock>>>(dev_a, dev_b, dev_c, N);
        
    hipError_t err1 = hipMemcpy(a, dev_a, N*sizeof(float), hipMemcpyDeviceToHost);
    printf(hipGetErrorString (err1));
    hipMemcpy(b, dev_b, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(c, dev_c, N*sizeof(float), hipMemcpyDeviceToHost);

    printf("\nRAM massives after CUDA kernel\n");
    printf("----------------------\n");
    printf("a\tb\tc\n");
    printf("----------------------\n");
    for(int i=0;i<N;i++)
    {        
        printf("%g\t%g\t%g\n", a[i], b[i], c[i]);
    }
    printf("----------------------\n");

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    free(a);
    free(b);
    free(c);

    return 0;
}