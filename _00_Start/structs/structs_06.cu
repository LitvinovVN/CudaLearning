/*
Добавить структуру Arrays1DList для хранения массива экземпляров структур Array1D
*/


#include <hip/hip_runtime.h>
#include <iostream>

struct Array1D
{
    int* arr;
    int  len;
};


__host__ __device__
void print_1D_array(int* arr, int n)
{
    printf("[");
    for(int i = 0; i < n-1; i++)
        printf("%d, ", arr[i]);
    printf("%d", arr[n-1]);
    printf("]\n");
}

__host__ __device__
void Array1D_print(Array1D* arr1D)
{
    printf("Array1D.len = %d\n", arr1D->len);
    printf("Array1D.arr = ");
    print_1D_array(arr1D->arr, arr1D->len);
}

__host__ __device__
void Array1D_print(Array1D arr1D)
{
    Array1D_print(&arr1D);
}

// Kernel
__global__ void add_M_Kernel(Array1D arr1D, int M)
{
    printf("\n--- START __global__ void add_M_Kernel(Array1D arr1D, int M) START ---\n");
    Array1D_print(&arr1D);
    for(int i=0; i<arr1D.len; i++)
        arr1D.arr[i] += M;
    Array1D_print(&arr1D);
    printf("--- END __global__ void add_M_Kernel(Array1D arr1D, int M) END ---\n\n");
}

Array1D Array1D_create_RAM(int numElements)
{
    int* array = (int*)malloc(numElements * sizeof(int));
    for(int i = 0; i < numElements; i++)
        array[i] = i;    
    
    Array1D array1D;
    array1D.len = numElements;
    array1D.arr = array;
    
    return array1D;
}

Array1D Array1D_create_GPU(int numElements)
{
    int* array = (int*)malloc(numElements * sizeof(int));
    for(int i = 0; i < numElements; i++)
        array[i] = i;    

    Array1D array1D_GPU;
    array1D_GPU.len = numElements;
    hipMalloc( &(array1D_GPU.arr), array1D_GPU.len * sizeof( array1D_GPU.arr ) );
    hipMemcpy(array1D_GPU.arr, array, array1D_GPU.len * sizeof(array1D_GPU.arr), hipMemcpyHostToDevice);

    free(array);

    return array1D_GPU;
}

Array1D Array1D_create_GPU(Array1D array1D_RAM)
{
    Array1D array1D_GPU;
    array1D_GPU.len = array1D_RAM.len;

    hipMalloc( &(array1D_GPU.arr), array1D_GPU.len * sizeof( array1D_GPU.arr ) );
    hipMemcpy(array1D_GPU.arr, array1D_RAM.arr, array1D_GPU.len * sizeof(array1D_GPU.arr), hipMemcpyHostToDevice);

    return array1D_GPU;
}

Array1D Array1D_copy_GPU_to_RAM(Array1D array1D_GPU)
{
    Array1D array1D_RAM = Array1D_create_RAM(array1D_GPU.len);
    hipMemcpy(array1D_RAM.arr, array1D_GPU.arr, array1D_GPU.len * sizeof(array1D_GPU.arr), hipMemcpyDeviceToHost);
    return array1D_RAM;
}

/// @brief Очищает видеопамять
/// @param array1D_GPU 
void Array1D_free_GPU(Array1D array1D_GPU)
{
    hipFree( array1D_GPU.arr );
}


///////////////// Структура Arrays1D ///////////////
struct Arrays1DList
{
    Array1D* arrays1D;
    int numElements;
};


////////////////////////////////////////////////////

int main()
{
    // Создаём в ОЗУ два экземпляра структуры Array1D
    Array1D array1D_RAM_01 = Array1D_create_RAM(10);
    Array1D array1D_RAM_02 = Array1D_create_RAM(5);

    // Создаём в ОЗУ экземпляр структуры Arrays1DList
    Arrays1DList arrays1DList_01;
    arrays1DList_01.numElements = 2;
    arrays1DList_01.arrays1D = (Array1D*)malloc(arrays1DList_01.numElements * sizeof(arrays1DList_01.arrays1D));
    arrays1DList_01.arrays1D[0] = array1D_RAM_01;
    arrays1DList_01.arrays1D[1] = array1D_RAM_02;
    Array1D_print(arrays1DList_01.arrays1D[0]);
    Array1D_print(arrays1DList_01.arrays1D[1]);   

    return 1;
}