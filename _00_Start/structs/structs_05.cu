/*
Добавить метод void Array1D_free_GPU(Array1D array1D_GPU)
*/


#include <hip/hip_runtime.h>
#include <iostream>

struct Array1D
{
    int* arr;
    int  len;
};

__host__ __device__
void print_1D_array(int* arr, int n)
{
    printf("[");
    for(int i = 0; i < n-1; i++)
        printf("%d, ", arr[i]);
    printf("%d", arr[n-1]);
    printf("]\n");
}

__host__ __device__
void Array1D_print(Array1D* arr1D)
{
    printf("Array1D.len = %d\n", arr1D->len);
    printf("Array1D.arr = ");
    print_1D_array(arr1D->arr, arr1D->len);
}

__host__ __device__
void Array1D_print(Array1D arr1D)
{
    Array1D_print(&arr1D);
}

// Kernel
__global__ void add_M_Kernel(Array1D arr1D, int M)
{
    printf("\n--- START __global__ void add_M_Kernel(Array1D arr1D, int M) START ---\n");
    Array1D_print(&arr1D);
    for(int i=0; i<arr1D.len; i++)
        arr1D.arr[i] += M;
    Array1D_print(&arr1D);
    printf("--- END __global__ void add_M_Kernel(Array1D arr1D, int M) END ---\n\n");
}

Array1D Array1D_create_RAM(int numElements)
{
    int* array = (int*)malloc(numElements * sizeof(int));
    for(int i = 0; i < numElements; i++)
        array[i] = i;    
    
    Array1D array1D;
    array1D.len = numElements;
    array1D.arr = array;
    
    return array1D;
}

Array1D Array1D_create_GPU(int numElements)
{
    int* array = (int*)malloc(numElements * sizeof(int));
    for(int i = 0; i < numElements; i++)
        array[i] = i;    

    Array1D array1D_GPU;
    array1D_GPU.len = numElements;
    hipMalloc( &(array1D_GPU.arr), array1D_GPU.len * sizeof( array1D_GPU.arr ) );
    hipMemcpy(array1D_GPU.arr, array, array1D_GPU.len * sizeof(array1D_GPU.arr), hipMemcpyHostToDevice);

    free(array);

    return array1D_GPU;
}

Array1D Array1D_create_GPU(Array1D array1D_RAM)
{
    Array1D array1D_GPU;
    array1D_GPU.len = array1D_RAM.len;

    hipMalloc( &(array1D_GPU.arr), array1D_GPU.len * sizeof( array1D_GPU.arr ) );
    hipMemcpy(array1D_GPU.arr, array1D_RAM.arr, array1D_GPU.len * sizeof(array1D_GPU.arr), hipMemcpyHostToDevice);

    return array1D_GPU;
}

Array1D Array1D_copy_GPU_to_RAM(Array1D array1D_GPU)
{
    Array1D array1D_RAM = Array1D_create_RAM(array1D_GPU.len);
    hipMemcpy(array1D_RAM.arr, array1D_GPU.arr, array1D_GPU.len * sizeof(array1D_GPU.arr), hipMemcpyDeviceToHost);
    return array1D_RAM;
}

/// @brief Очищает видеопамять
/// @param array1D_GPU 
void Array1D_free_GPU(Array1D array1D_GPU)
{
    hipFree( array1D_GPU.arr );
}

int main()
{
    Array1D array1D_RAM = Array1D_create_RAM(10);
    Array1D array1D_GPU = Array1D_create_GPU(array1D_RAM);
        
    add_M_Kernel<<< 1, 1 >>>( array1D_GPU, 5 );
    
    Array1D array1D_RAM_result = Array1D_copy_GPU_to_RAM(array1D_GPU);
    Array1D_print(array1D_RAM_result);

    Array1D_free_GPU(array1D_GPU);

    return 1;
}