
#include <hip/hip_runtime.h>
#include <iostream>

__device__ int getGlobalIndex()
{
    //Индекс текущего блока в гриде
    int blockIndex = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.y*gridDim.x;
    //Индекс треда внутри текущего блока
    int threadIndex = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.y*blockDim.x;

    //глобальный индекс нити
    int i = blockIndex*blockDim.x*blockDim.y*blockDim.z + threadIndex; 

    return i;
}

__global__ void cuda_threadIdx(float* A, float* B, float* C, int size){
    int i = getGlobalIndex();
    
    C[i] = A[i] + B[i];
}

int main() {
    // Размерность массива
    int N = 10;

    // Размер массива, байт
    int dataSize = N * sizeof(float);
    printf("dataSize = %d bytes\n", dataSize);

    // Выделение памяти в ОЗУ
    float *a = (float*)malloc(dataSize);
    float *b = (float*)malloc(dataSize);
    float *c = (float*)malloc(dataSize);

    printf("RAM massives initialization\n");
    printf("a\tb\tc\n");
    for(int i=0;i<N;i++)
    {
        a[i] = i;
        b[i] = 0.2*i;
        c[i] = 0;
        printf("%g\t%g\t%g\n", a[i], b[i], c[i]);
    }

    // Выделение памяти в GPU
    float *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, dataSize);
    hipMalloc((void**)&dev_b, dataSize);
    hipMalloc((void**)&dev_c, dataSize);
   
    
    hipMemcpy(dev_a, a, dataSize, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, dataSize, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, dataSize, hipMemcpyHostToDevice);

    int numBlocks = 2;
    dim3 threadsPerBlock(N/numBlocks);

    cuda_threadIdx<<<numBlocks,threadsPerBlock>>>(dev_a, dev_b, dev_c, N);
        
    hipError_t err1 = hipMemcpy(a, dev_a, dataSize, hipMemcpyDeviceToHost);
    printf(hipGetErrorString (err1));
    hipMemcpy(b, dev_b, dataSize, hipMemcpyDeviceToHost);
    hipMemcpy(c, dev_c, dataSize, hipMemcpyDeviceToHost);

    printf("\nRAM massives after CUDA kernel\n");
    printf("----------------------\n");
    printf("a\tb\tc\n");
    printf("----------------------\n");
    for(int i=0;i<N;i++)
    {        
        printf("%g\t%g\t%g\n", a[i], b[i], c[i]);
    }
    printf("----------------------\n");

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    free(a);
    free(b);
    free(c);

    return 0;
}