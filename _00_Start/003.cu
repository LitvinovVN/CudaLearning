// Задача 003. Передать в CUDA-ядро число типа float.
// Увеличить его на 1.5. Скопировать измененное значение из GPU в ОЗУ.
// Вывести измененное значение переменной в консоль.
// Запуск:
// nvcc 003.cu
// ./a


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void cuda_hello(float* dev_varFloat){    
    printf("cuda_hello START: varFloat = %f\n", *dev_varFloat);
    *dev_varFloat += 1.5;
    printf("cuda_hello END: varFloat = %f\n", *dev_varFloat);
}

int main() {
    float varFloat = -43.0123456789;
    printf("main START: varFloat = %f\n", varFloat);

    float* dev_varFloat;
    hipMalloc((void**)&dev_varFloat, sizeof(float));
    hipMemcpy(dev_varFloat, &varFloat, sizeof(float), hipMemcpyHostToDevice);

    cuda_hello<<<1,1>>>(dev_varFloat);
    hipMemcpy(&varFloat, dev_varFloat, sizeof(float), hipMemcpyDeviceToHost);
    
    printf("main END: varFloat = %f\n", varFloat);

    return 0;
}