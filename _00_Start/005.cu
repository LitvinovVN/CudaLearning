// Задача 005.
// Создать структуру, содержащую три числа типов bool, int, float.
// Скопировать структуру из ОЗУ в GPU.
// Передать в CUDA-ядро структуру. 
// Вывести значения элементов структуры в консоль из CUDA-ядра.
// Запуск:
// nvcc 005.cu
// ./a


#include <hip/hip_runtime.h>
#include <iostream>

typedef struct 
{
    bool varBool;
    int varInt;
    float varFloat;
} my_struct;

__global__ void cuda_struct_print(my_struct* dev_my_struct){ 
    printf("----- cuda_struct_print START -----\n");
    printf("dev_my_struct->varInt = %d\n", dev_my_struct->varBool);
    printf("dev_my_struct->varInt = %d\n", dev_my_struct->varInt);
    printf("dev_my_struct->varInt = %f\n", dev_my_struct->varFloat);
    printf("----- cuda_struct_print END -----\n");
}

int main() {    
    my_struct* ram_my_struct = (my_struct*)malloc(sizeof(my_struct));
    ram_my_struct->varBool = true;
    ram_my_struct->varInt = -5;
    ram_my_struct->varFloat = 25.5;

    my_struct* dev_my_struct;
    hipMalloc((void**)&dev_my_struct, sizeof(my_struct));
    hipMemcpy(dev_my_struct, ram_my_struct, sizeof(my_struct), hipMemcpyHostToDevice);

    cuda_struct_print<<<1,1>>>(dev_my_struct);
    
    return 0;
}