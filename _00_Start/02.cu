#include "hip/hip_runtime.h"
#include <iostream>

__global__ void cuda_threadIdx(float* A, float* B, float* C, int size){
    printf("Hello World from GPU!\n");
    printf("A[2]=%g\n", A[2]);
    int i = 2;// threadIdx.x;
    //C[i] = A[i] + B[i]; 
    A[2] = 10;
    printf("A[2]=%g\n", A[2]);
}

int main() {
    // Размерность массива
    int N = 10;

    // Выделение памяти в ОЗУ
    float *a = (float*)malloc(N*sizeof(float));
    float *b = (float*)malloc(N*sizeof(float));
    float *c = (float*)malloc(N*sizeof(float));

    // Выделение памяти в GPU
    float *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, N*sizeof(float));
    hipMalloc((void**)&dev_b, N*sizeof(float));
    hipMalloc((void**)&dev_c, N*sizeof(float));

    printf("RAM massives initialization\n");
    for(int i=0;i<N;i++)
    {
        a[i] = i;
        b[i] = -i;
        c[i] = 0.01*i;
        printf("%g %g %g \n", a[i], b[i], c[i]);
    }
    
    hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    int numBlocks = 3;
    dim3 threadsPerBlock(N);
    //cuda_threadIdx<<<numBlocks,threadsPerBlock>>>(dev_a, dev_b, dev_c, N);
    cuda_threadIdx<<<1,1>>>(dev_a, dev_b, dev_c, N);
    
    hipError_t err1 = hipMemcpy(a, dev_a, N*sizeof(float), hipMemcpyDeviceToHost);
    printf(hipGetErrorString (err1));
    hipMemcpy(b, dev_b, N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(c, dev_c, N*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    printf("\nRAM massives after CUDA kernel\n");
    for(int i=0;i<N;i++)
    {
        a[i] = i;
        b[i] = -i;
        c[i] = 0.01*i;
        printf("%g %g %g \n", a[i], b[i], c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}