#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// ������������ �����
const int Nx = 10;
const int Ny = 20;
const int Nz = 30;
const int Nblocks = Nx * Ny * Nz;
// ������������ ����� � ����� (������ � 1 �����)
const int Ndx = 4;
const int Ndy = 3;
const int Ndz = 2;
const int NtreadsInBlock = Ndx * Ndy * Ndz;

const int N = Nblocks * NtreadsInBlock;

hipError_t addWithCuda(double* c, const double* a, const double* b, unsigned int size);

__global__ void addKernel(double* c, const double* a, const double* b)
{
    int indexXY = threadIdx.x + threadIdx.y * blockDim.x;//����� ������ � ������� ��������� XY �������� �����
    int NumXY = blockDim.x * blockDim.y;//����� ������� � ������� ��������� XY �������� �����
    int indexXYZ = indexXY + NumXY * threadIdx.z;//����� ������ � ������� �����

    //printf("indexXYZ = %d\n", indexXYZ);

    int indexXYBlock = blockIdx.x + blockIdx.y * gridDim.x;//����� ����� � ������� ��������� XY �����
    int NumXYBlock = gridDim.x * gridDim.y;//����� ������ � ������� ��������� XY �����
    int indexXYZBlock = indexXYBlock + NumXYBlock * blockIdx.z;//����� ����� � �����

    int index = indexXYZBlock * blockDim.x * blockDim.y * blockDim.z + indexXYZ;//���������� ������ ����

    if (blockIdx.x == 1 && blockIdx.y == 2 && blockIdx.z == 3 && threadIdx.x == 3 && threadIdx.y == 2 && threadIdx.z == 1)
    {
        printf("gridDim.x = %d, gridDim.y = %d, gridDim.z = %d\n",gridDim.x, gridDim.y, gridDim.z);
        printf("blockDim.x = %d, blockDim.y = %d, blockDim.z = %d\n", blockDim.x, blockDim.y, blockDim.z);        
        printf("indexXYZ = %d\n", indexXYZ);                
        printf("indexXYZBlock = %d\n", indexXYZBlock);
        printf("index = %d\n", index);
    }


    //������ �������� ����� � �����
    int blockIndex = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.y * gridDim.x;
    //������ ����� ������ �������� �����
    int ThreadIndex = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;

    //���������� ������ ����
    int GlobalThreadIndex = blockIndex * blockDim.x * blockDim.y * blockDim.z + ThreadIndex;

    //printf("block: (%d,%d,%d) threads: (%d,%d,%d) blockIndex = %d ThreadIndex = %d GlobalThreadIndex= %d\n ", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z, blockIndex, ThreadIndex, GlobalThreadIndex);

    /*if (index != GlobalThreadIndex)
        printf("False\n");*/

    c[GlobalThreadIndex] = a[GlobalThreadIndex] + b[GlobalThreadIndex];
}

int main()
{    
    double* a = new double[N];
    double* b = new double[N];
    double* c = new double[N];
    
    for (size_t z = 0; z < Nz; z++)
    {
        for (size_t y = 0; y < Ny; y++)
        {
            for (size_t x = 0; x < Nx; x++)
            {
                int i = x + y * Nx + z * Nx * Ny;
                a[i] = i;
                b[i] = 2 * i;
            }
        }
    }



    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{%.1lf,%.1lf,%.1lf,%.1lf,%.1lf} + {%.1lf,%.1lf,%.1lf,%.1lf,%.1lf} = {%.1lf,%.1lf,%.1lf,%.1lf,%.1lf}\n",
        a[0], a[1], a[2], a[3], a[4], b[0], b[1], b[2], b[3], b[4], c[0], c[1], c[2], c[3], c[4]);

    for (size_t i = 0; i < N; i++)
    {
        if (a[i] + b[i] == c[i])
            printf("True!!!!!");
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(double *c, const double *a, const double *b, unsigned int size)
{
    double *dev_a = 0;
    double *dev_b = 0;
    double *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.    
    dim3 NxNyNz(Nx, Ny, Nz);
    dim3 NdxNdyNdz(Ndx, Ndy, Ndz);
    addKernel<<<NxNyNz, NdxNdyNdz >>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
