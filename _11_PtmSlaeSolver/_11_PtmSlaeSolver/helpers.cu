#include "hip/hip_runtime.h"
#ifndef HELPER_FILE
#define HELPER_FILE

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <time.h>
#include "locale.h"
#include <malloc.h>
#include <stdlib.h>
#include <vector>
#include <array>
#include <thread>

using ::std::thread;
using ::std::array;
using ::std::vector;
using ::std::cout;
using ::std::endl;
using ::std::ref;


/// <summary>
/// ����������� � ��������� ������������
/// </summary>
struct Dim3d {
    size_t x;
    size_t y;
    size_t z;
};

/// <summary>
/// �������� ��������� �����
/// </summary>
struct GridFragment3d {
    Dim3d dimensions;
};

/// <summary>
/// ��������� �����
/// </summary>
struct Grid3d {
    Dim3d dimensions{};

    Grid3d(int x, int y, int z) {
        dimensions.x = x;
        dimensions.y = y;
        dimensions.z = z;
    }

    /// <summary>
    /// ������� � ������� ��������� ��������� �����
    /// </summary>
    __device__ __host__
    void print_dimensions()
    {
        printf("����������� ��������� �����: {%d, %d, %d}\n", dimensions.x, dimensions.y, dimensions.z);
    }
};

/// <summary>
/// ���������� �������� � �������
/// </summary>
inline void ShowSystemProperties() {
    std::cout << std::endl;
    std::cout << "---------------- �������� � ������� -----------------" << std::endl;
    std::cout << "���������� ��������� ������� (���� CPU):" << std::thread::hardware_concurrency() << std::endl;
    std::cout << "-----------------------------------------------------" << std::endl;
}

//////////////////////////////////////////////
/// <summary>
/// ���������� ��������� �������������
/// </summary>
inline void ShowVideoadapterProperties() {
    hipDeviceProp_t prop;
    int count;
    hipGetDeviceCount(&count);
    for (size_t i = 0; i < count; i++)
    {
        hipGetDeviceProperties(&prop, i);
        printf("������������ ����������:        %s\n", prop.name);
        printf("�������������� �����������:     %d.%d\n", prop.major, prop.minor);
        printf("�������� �������:               %d ���\n", prop.clockRate / 1000);
        printf("���������� ����������� (���.): ");
        if (prop.deviceOverlap)
        {
            printf("���������\n");
        }
        else
        {
            printf("���������\n");
        }
        printf("����-��� ���������� ����: ");
        if (prop.kernelExecTimeoutEnabled)
        {
            printf("�������\n");
        }
        else
        {
            printf("��������\n");
        }
        printf("���������� ����������� DMA �������: %d (1: ����������� ������ + ����, 2: ����������� ������ up + ����������� ������ down + ����)\n", prop.asyncEngineCount);

        printf("------------ ���������� � ������ ---------------\n");
        printf("����� ���������� ������:        %ld ����\n", prop.totalGlobalMem);
        printf("����� ����������� ������:       %ld ����\n", prop.totalConstMem);

        printf("------------ ���������� � ����������������� ---------------\n");
        printf("���������� �����������������:   %d\n", prop.multiProcessorCount);
        printf("���������� �������������� ������ �� 1 ����:   %d ����\n", prop.sharedMemPerBlock);
        printf("���������� �������������� ������ �� 1 ���������������:   %ld ����\n", prop.sharedMemPerMultiprocessor);
        printf("���������� 32�-������ ��������� �� 1 ����:   %d ����\n", prop.regsPerBlock);
        printf("������ warp'�:                  %d\n", prop.warpSize);
        printf("������������ ���������� ����� � �����: %d\n", prop.maxThreadsPerBlock);
        printf("������������ ���������� ����� � �����: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("������������ ������� �����: (%ld, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    }
}


inline __device__ int returnsPlus2(int num)
{
    return num + 2;
}



inline __global__ void addKernel(int* c, const int* a, const int* b, Grid3d* g)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

    //g->print_dimensions();
    int n = returnsPlus2(5);
    printf("\n%d\n", n);
}

// Helper function for using CUDA to add vectors in parallel.
inline hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    Grid3d g(10,20,30);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b, &g);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

inline int addWithCudaStart() {
#pragma region addWithCuda
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
#pragma endregion
}

#endif