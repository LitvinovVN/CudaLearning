#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#define Nx 20 // ������ ���� �� x
#define Ny 20 // ������ ���� �� y
#define Nz 10 // ����������� �� z
#define N (Nx*Ny*Nz)
#define hx 1 // ��� �� x
#define hy 1 // ��� �� y
#define hz 1 // ��� �� z
#define lt 10 // �����
#define ht 0.1 // ��� �� �������
#define sigma 0.5
#define err 0.000000001

#define blocks 1
#define threads ((Nx*Ny + blocks - 1) / blocks)

double O[N], // ������� ������������� ������
v[N], // ��������� ������� ��������
u[N], // ��������� ������� ��������
w[N], // ��������� ������� ��������
mu[N], // ���������� �������� 
Cn[N], // ������� ����
B1[N],
B2[N],
B3[N],
B4[N],
B5[N],
B6[N],
A[N],
F[N]; // �-� ��������

__host__ __device__ void printMatrix(double arr[N]) {
    for (int i = 0; i < Nx; i++) {
        for (int j = 0; j < Ny; j++) {
            printf("%f ", arr[i * Nx + j]);
        }
        printf("\n");
    }
}

__host__ __device__ void printVector(int arr[Nx]) {
    for (int i = 0; i < Nx; i++) {
        printf("%d ", arr[i]);
    }
    printf("\n");
}

void writeToFile(double arr[N], const char* fileName) {
    FILE* f = fopen(fileName, "w+t");
    if (f) {
        for (int k = 0; k < Nz; k++) {
            for (int i = 0; i < Nx; i++) {
                for (int j = 0; j < Ny; j++) {
                    int m0 = i + j * Nx + k * Nx * Ny;
                    fprintf(f, "%f ", arr[m0]);
                }
                fprintf(f, "\n");
            }
        }
    }
    fclose(f);
}

__device__
void calc(double* O, double* u, double* v, double* w, double* mu, double* C, // arrays
    double* A, double* B1, double* B2, double* B3, // values
    double* B4, double* B5, double* B6, double* F, int m0) {

    double B51, B61, B7, B8, B9, B10, B11;

    int m1 = m0 + 1;
    int m2 = m0 - 1;
    int m3 = m0 + Nx;
    int m4 = m0 - Nx;
    int m5 = m0 + Nx * Ny;
    int m6 = m0 - Nx * Ny;
    int m24 = m4 - 1;
    int m26 = m2 - Nx * Ny;
    int m46 = m4 - Nx * Ny;
    int m246 = m24 - Nx * Ny;

    double q1 = (O[m0] + O[m4]  + O[m6]  + O[m46])  / 4; //������������� ������� D
    double q2 = (O[m2] + O[m24] + O[m26] + O[m246]) / 4;
    double q3 = (O[m0] + O[m2]  + O[m6]  + O[m26])  / 4;
    double q4 = (O[m4] + O[m24] + O[m46] + O[m246]) / 4;
    double q5 = (O[m0] + O[m2]  + O[m4]  + O[m24])  / 4;
    double q6 = (O[m6] + O[m26] + O[m46] + O[m246]) / 4;
    double q0 = (q1 + q2) / 2;

    //���������� ����� ��� ��������-��������� � ������������ �����.
    // ??????????????????????????????????????????????????????????????
    *B1 = q1 * (-(u[m1] + u[m0]) / (4 * hx) + (mu[m1] + mu[m0]) / (2 * hx * hx));
    *B2 = q2 * ( (u[m2] + u[m0]) / (4 * hx) + (mu[m2] + mu[m0]) / (2 * hx * hx));
    *B3 = q3 * (-(v[m3] + v[m0]) / (4 * hy) + (mu[m3] + mu[m0]) / (2 * hy * hy));
    *B4 = q4 * ( (v[m4] + v[m0]) / (4 * hy) + (mu[m4] + mu[m0]) / (2 * hy * hy));
    *B5 = q5 * (-(w[m5] + w[m0]) / (4 * hz) + (mu[m5] + mu[m0]) / (2 * hz * hz));
    *B6 = q6 * ( (w[m6] + w[m0]) / (4 * hz) + (mu[m6] + mu[m0]) / (2 * hz * hz));

    B61 = (1 - sigma) * (*B1);
    B7  = (1 - sigma) * (*B2);
    B8  = (1 - sigma) * (*B3);
    B9  = (1 - sigma) * (*B4);
    B10 = (1 - sigma) * (*B5);
    B11 = (1 - sigma) * (*B6);

    *B1 = sigma * (*B1);
    *B2 = sigma * (*B2);
    *B3 = sigma * (*B3);
    *B4 = sigma * (*B4);
    *B5 = sigma * (*B5);
    *B6 = sigma * (*B6);

    *A = q0 / ht + (*B1) + (*B2) + (*B3) + (*B4) + (*B5) + (*B6);
    B51 = q0 / ht - B61 - B7 - B8 - B9 - B10 - B11;

    *F = B51 * C[m0] + B61 * C[m1] + B7 * C[m2] + B8 * C[m3] + B9 * C[m4] + B10 * C[m5] + B11 * C[m6];
}

__device__ int max_found = 0;

__global__
void processCalculating(double* O, double* u, double* v, double* w, double* mu, double* C) {
    double A[Nz], B1[Nz], B2[Nz], B3[Nz], B4[Nz], B5[Nz], B6[Nz], F[Nz];
    double t = 0;

    int i = blockIdx.x * blockDim.x + threadIdx.x + 1; // +1 �.� �� �������� � 1-�� �������

    int cx = i % Nx;// ���������� �� x
    int cy = i / Nx;// ���������� �� y
    //if(cx == 19) printf("cx = %d, cy = %d\n", cx, cy);

    //if (i == 1) printf("\nlockIdx: %d %d %d \n", blockIdx.x, blockIdx.y, blockIdx.z);
    //printf("ThreadIdx: %d %d %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
    /*if (i == 1) {
        for (size_t i = 0; i < N; i++)
        {
            printf("i = %d; O = %lf; u = %lf; v = %lf; w = %lf; mu = %lf\n", i, O[i], u[i], v[i], w[i], mu[i]);
        }
    }*/

    if (i >= Nx*Ny - 1) return;

    do {
        // ������������ ����� ��������
        for (int k = 1; k < Nz - 1; k++) {
            calc(O, u, v, w, mu, C, &A[k], &B1[k], &B2[k],
                &B3[k], &B4[k], &B5[k], &B6[k], &F[k], i + k * Nx*Ny);

            if (B3[k] > 0 + err || B3[k] < 0 - err) {
                printf("k = %d; A = %lf; B1 = %lf; B2 = %lf; B3 = %lf; B4 = %lf; B5 = %lf; B6 = %lf\n", k, A[k], B1[k], B2[k], B3[k], B4[k], B5[k], B6[k]);
            }
        }

        // ���� ������ �� ��������� ������������ ������
        do {
            atomicCAS(&max_found, 1, 0);
            for (int j = 1; j < 2 * Ny - 3; j++) {
                // ������������ ����� ��������
                int m0 = i + (j - i + 1) * Nx, m1, m2, m3, m4;
                int k = m0 / Nx;
                double w = C[m0];

                if (i > j || (j - i) >= (Nx - 2)) goto l_break;

                m1 = m0 + 1;
                m2 = m0 - 1;
                m3 = m0 + Nx;
                m4 = m0 - Nx;

                C[m0] = (F[k] + B1[k] * C[m1] + B2[k] * C[m2] + B3[k] * C[m3] + B4[k] * C[m4]) / A[k];

                w = fabs(w - C[m0]);
                if (w >= err) {
                    atomicExch(&max_found, 1);
                }
            l_break:
                __syncthreads();
            }
        } while (max_found != 0);

        t += ht;

    } while (t < lt);
}

int main(int argc, char const* argv[]) {
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

    printf("Starting...\n");

    for (int k = 0; k < Nz; k++) {
        for (int i = 0; i < Nx; i++) {
            for (int j = 0; j < Ny; j++) {
                int m0 = i + j * Nx + k * Nx * Ny;
                Cn[m0] = 0;
                O[m0] = 0.1;
                mu[m0] = 0.2;
                u[m0] = 0.3;
                v[m0] = 0.4;
                w[m0] = 0.1;
            }
        }
    }

    printf("Initial values filled\n");

    for (int k = 0; k < Nz / 4; k++) {
        for (int i = 1; i < Nx / 4; i++) {
            for (int j = 1; j < Ny / 4; j++) {
                int m0 = i + j * Nx + k * Nx * Ny;
                Cn[m0] = 1;
            }
        }
    }

    printf("Cn values filled\n");

    writeToFile(Cn, "start_cuda.txt");

    double* c_O, * c_u, * c_v, * c_w, * c_mu, * c_C;

    // alloc all arrays
    hipMalloc(&c_O, N * sizeof(double));
    hipMalloc(&c_u, N * sizeof(double));
    hipMalloc(&c_v, N * sizeof(double));
    hipMalloc(&c_w, N * sizeof(double));
    hipMalloc(&c_mu, N * sizeof(double));
    hipMalloc(&c_C, N * sizeof(double));

    printf("Cuda values allocated\n");

    // copy static values
    hipMemcpy(c_O, O, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(c_u, u, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(c_v, v, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(c_w, w, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(c_mu, mu, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(c_C, Cn, N * sizeof(double), hipMemcpyHostToDevice);

    printf("Cuda values copied\n");

    printf("Trying to calling kernel...\n");
    processCalculating << <blocks, threads >> > (c_O, c_u, c_v, c_w, c_mu, c_C);
    printf("Called:-)\n");
    hipDeviceSynchronize();
    printf("Synced:-)\n");

    hipMemcpy(Cn, c_C, N * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(c_O);
    hipFree(c_u);
    hipFree(c_v);
    hipFree(c_w);
    hipFree(c_mu);
    hipFree(c_C);

    writeToFile(Cn, "result_cuda.txt");

    return 0;
}