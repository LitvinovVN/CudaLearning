#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "locale.h"
#include <malloc.h>
#include <stdlib.h>
using namespace std;

//////////////// ���������� ////////////////
// GT710 2Gb, Core i5-7400 3GHz, 16 Gb ���
// 100x100x160
// nevyazkaKernelTime = 2019 ms
// nevyazkaGreaterEpsKernel = 0.9 ms
// ptmKernel1. i = 3;   gpuTime = 86.5 ms
// ptmKernel1. i = 182; gpuTime = 92.09 ms
// ptmKernel1. i = 356; gpuTime = 86.9 ms
// ptmKernel2 = 39647 ms
// awrRrKernel + RwRw + Aww + ww = 3029 ms
// uKernel = 281 ms
////////////////////////////////////////////
// GTX 750 Ti, Core i5-6600 3.3GHz, 32 Gb ���
// 100x100x160
// nevyazkaKernelTime = 195 ms
// nevyazkaGreaterEpsKernel = 1.9 ms
// ptmKernel1. i = 3;   gpuTime = 15.3 ms
// ptmKernel1. i = 182; gpuTime = 15.7 ms
// ptmKernel1. i = 356; gpuTime = 15.0 ms
// ptmKernel2 = 5417 ms
// awrRrKernel + RwRw + Aww + ww = 319 ms
// uKernel = 137.7 ms
////////////////////////////////////////////

#define BLOCK_SIZE 256

#define GridNx 100 // ����������� ��������� ����� �� ��� x
#define GridNy 100 // ����������� ��������� ����� �� ��� y
#define GridNz 160 // ����������� ��������� ����� �� ��� z
#define GridN GridNx*GridNy*GridNz // ��������� ����� ����� ��������� �����
#define GridXY GridNx * GridNy // ����� ����� � ��������� XY, �.�. � ����� ���� �� Z

#define CudaCoresNumber 192 // ���������� ���� cuda (https://geforce-gtx.com/710.html - ��� GT710, ��� ������ ���������� ���������� ��������)

#define EPS 0.001

void Print3dArray(int* host_c);
void Print3dArrayDouble(double* host_c);
double Reduce(double* data, long n);

void PtmTest();
void ReductionTest();

#pragma region Kernels

__global__ void uKernel(double* u, double* r, long size, double tay)
{
    const size_t threadX = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t threadY = blockDim.y * blockIdx.y + threadIdx.y;

    if (threadX >= GridNx - 1 || threadY >= GridNy - 1 || threadX == 0 || threadY == 0)
        return;

    const size_t idx = threadY * GridNx + threadX;
        
    long nodeIndex = idx;
    for (size_t z = 1; z < GridNz - 1; z++)
    {
        long m0 = nodeIndex + z * GridXY;        

        if (nodeIndex < size)
        {
            u[m0] = u[m0] + tay * r[m0];
        }
    }
}

__global__ void sumElFromN1ToN2Kernel(double* sum, double* data, long N1, long N2)
{
    *sum = 0;
    if (threadIdx.x == 0)
    {
        for (size_t i = N1; i <= N2; i++)
        {
            *sum += data[i];
            //printf("sumElFromN1ToN2Kernel, 41: *sum = %lf", *sum);
        }        
    }
}

__global__ void reduceKernel(double* inData, double* outData)
{
    __shared__ double data[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;

    // �������� ����� ������ ���� ��������� � ����������� ������
    data[tid] = inData[i] + inData[i + blockDim.x];
    
    __syncthreads();  // ��������� �������� ������

    for (int s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (tid < s)
        {
            data[tid] += data[tid + s];
        }

        __syncthreads();
    }

    if (tid < 32)  // ���������� ��������� ��������
    {
        data[tid] += data[tid + 32];
        data[tid] += data[tid + 16];
        data[tid] += data[tid + 8];
        data[tid] += data[tid + 4];
        data[tid] += data[tid + 2];
        data[tid] += data[tid + 1];
    }

    if (tid == 0)  // ��������� ����� ��������� �����
    {
        outData[blockIdx.x] = data[0];        
    }
}

#pragma endregion Kernels

/// <summary>
/// ������������ ������� ���������
/// </summary>
/// <param name="data"></param>
/// <param name="n"></param>
/// <returns></returns>
double Reduce(double* data, long n)
{
    double res = 0;

    double* sums = NULL;
    int numBlocks = n / 512;

    //tex:
    // ��������� �������� � ������ ������� �� $$numBlocks \times 512$$ �� $$n - 1$$ 
    double sumRight = 0;
    long N1 = numBlocks * 512;
    long N2 = n - 1;
    //printf("N1 = %d\n", N1);
    //printf("N2 = %d\n", N2);
    
    double* dev_sumRight = NULL;
    hipMalloc((void**)&dev_sumRight, sizeof(double));
    sumElFromN1ToN2Kernel <<< 1, 1 >> > (dev_sumRight, data, N1, N2);
    hipMemcpy(&sumRight, dev_sumRight, sizeof(double), hipMemcpyDeviceToHost);
    //printf("sumRight = %lf\n", sumRight);
    
    res += sumRight;

    // �������� ������ ��� ������ ���� ������
    hipMalloc( (void**) &sums, numBlocks * sizeof(double));

    // �������� ��������� ��������, ������� ����� ��� ������� ����� � ������ sums
    reduceKernel << <dim3(numBlocks), dim3(BLOCK_SIZE) >> > (data, sums);

    // ���������� ������ ���� ��� ������
    if (numBlocks > BLOCK_SIZE)
    {
        res = Reduce(sums, numBlocks);
    }
    else
    {
        double* sumsHost = new double[numBlocks];

        hipMemcpy(sumsHost, sums, numBlocks * sizeof(double), hipMemcpyDeviceToHost);

        for (int i = 0; i < numBlocks; i++)
        {
            res += sumsHost[i];
        }

        delete[] sumsHost;
    }

    hipFree(sums);
    return res;
}


/// <summary>
/// ���������� ��������� �������������
/// </summary>
void ShowVideoadapterProperties() {
    hipDeviceProp_t prop;
    int count;
    hipGetDeviceCount(&count);
    for (size_t i = 0; i < count; i++)
    {
        hipGetDeviceProperties(&prop, i);
        printf("������������ ����������:        %s\n", prop.name);
        printf("�������������� �����������:     %d.%d\n", prop.major, prop.minor);
        printf("���������� �����������������:   %d\n", prop.multiProcessorCount);
        printf("������ warp'�:                  %d\n", prop.warpSize);
    }

    printf("���������� ���� cuda: %d (�������� �� ������������ � �������������)\n", CudaCoresNumber);
}

/// <summary>
/// ���������� ��������� �����
/// </summary>
void ShowGridProperties()
{
    printf("\n--------------�������������� ��������� �����----------------\n");
    printf("����������� ��������� ����� �� ��� x:                %d\n", GridNx);
    printf("����������� ��������� ����� �� ��� y:                %d\n", GridNy);
    printf("����������� ��������� ����� �� ��� z:                %d\n", GridNz);
    printf("��������� ����� ����� ��������� �����:               %d\n", GridN);
    printf("����� ����� � ��������� XY, �.�. � ����� ���� �� Z:  %d\n", GridXY);
    printf("----------------------------------------------------------\n");
}



int main()
{
    // ��������� ��������� ��������� � �������
    setlocale(LC_CTYPE, "rus");
    // ����������� ���������� ����������
    ShowVideoadapterProperties();
    // ����������� ���������� �����
    ShowGridProperties();
    // ���� ��������� ����������
    PtmTest();
    // ���� �������� �������
    // ReductionTest();

    return 0;
}

/// <summary>
/// ������������� ������� �� GPU
/// </summary>
/// <param name="c"></param>
/// <param name="size"></param>
/// <returns></returns>
__global__ void initVectorInGpuKernel(int* c, unsigned int size)
{
    // Compute the offset in each dimension
    const size_t offsetX = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t offsetY = blockDim.y * blockIdx.y + threadIdx.y;
    const size_t offsetZ = blockDim.z * blockIdx.z + threadIdx.z;

    // Make sure that you are not actually outs
    if (offsetX >= GridNx || offsetY >= GridNy || offsetZ >= GridNz)
        return;

    // Compute the linear index assuming that X,Y then Z memory ordering
    const size_t idx = offsetZ * GridNx * GridNy + offsetY * GridNx + offsetX;

    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0 && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0)
    {
        printf("\n--------------initVectorInGpuKernel-------------------\n");
        printf("threadIdx.x = %d, threadIdx.y = %d, threadIdx.z = %d\n", threadIdx.x, threadIdx.y, threadIdx.z);
        printf("blockIdx.x = %d,  blockIdx.y = %d,  blockIdx.z = %d\n", blockIdx.x, blockIdx.y, blockIdx.y);
        printf("blockDim.x = %d,  blockDim.y = %d , blockDim.z = %d\n", blockDim.x, blockDim.y, blockDim.z);
        printf("offsetX = %d,     offsetY = %d,     offsetZ = %d\n", offsetX, offsetY, offsetZ);
        printf("idx = %d\n", idx);
        printf("\n-----------initVectorInGpuKernel (end)--------------\n");
    }


    long nodeIndex = idx;
    for (size_t z = 0; z < GridNz; z++)
    {
        if (idx < size)
        {
            c[nodeIndex] = nodeIndex;
        }
        nodeIndex += GridNx * GridNy;
    }
}

/// <summary>
/// ������ �������
/// </summary>
__global__ void nevyazkaKernel(double* r, double* c0, double* c1, double* c2, double* c3, double* c4, double* c5, double* c6, double* f, double* u, unsigned int size)
{
    const size_t threadX = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t threadY = blockDim.y * blockIdx.y + threadIdx.y;

    if (threadX >= GridNx-1 || threadY >= GridNy-1 || threadX == 0 || threadY == 0)
        return;

    const size_t idx = threadY * GridNx + threadX;       


    long nodeIndex = idx;
    for (size_t z = 1; z < GridNz-1; z++)
    {
        long m0 = nodeIndex + z * GridXY;
        long m1 = m0 + 1;
        long m2 = m0 - 1;
        long m3 = m0 + GridNx;
        long m4 = m0 - GridNx;
        long m5 = m0 + GridXY;
        long m6 = m0 - GridXY;

        if (idx < size)
        {
            r[m0] = f[m0] - c0[m0] * u[m0] + (c1[m0] * u[m1] + c2[m0] * u[m2] + c3[m0] * u[m3] + c4[m0] * u[m4] + c5[m0] * u[m5] + c6[m0] * u[m6]);
            //printf("r[%d] = %lf; \n",m0, r[m0]);
        }        
    }    
}

__global__ void nevyazkaGreaterEpsKernel(int* isGreater, double* r, unsigned int size, double eps)
{
    //printf("----!!!!!!!!!! 170 !!!!!!!! isGreater = %d--------\n", *isGreater);
    if (*isGreater > 0)
    {
        //printf("----!!!!!!!!!! 172 !!!!!!!! isGreater = %d| isGreater > 0 ---> return; --------\n", *isGreater);
        return;
    }
    
    const size_t threadX = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t threadY = blockDim.y * blockIdx.y + threadIdx.y;

    if (threadX >= GridNx - 1 || threadY >= GridNy - 1 || threadX == 0 || threadY == 0)
        return;

    const size_t idx = threadY * GridNx + threadX;
        
    long nodeIndex = idx;
    for (size_t z = 1; z < GridNz - 1; z++)
    {
        long m0 = nodeIndex + z * GridXY;        
        
        if (r[m0] > eps && *isGreater == 0)
        {            
            atomicExch(isGreater, 1);
            //printf("----!!!!!!!!!--- 199 ---!!!!!!!!!r[%d] = %lf; isGreater = %d--------\n", m0, r[m0], *isGreater);
            return;
        }
    }
}

/// <summary>
/// ���, ������ �� 0 �� Nx+Ny+Nz
/// </summary>
/// <param name="c"></param>
/// <param name="size">���-�� ��������� ������� s</param>
/// <param name="s">i+j+k</param>
/// <returns></returns>
__global__ void ptmKernel1(double* r, double* c0, double* c2, double* c4, double* c6, unsigned int size, int s, double omega)
{
    // Compute the offset in each dimension
    const size_t threadX = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t threadY = blockDim.y * blockIdx.y + threadIdx.y;    

    // Make sure that you are not actually outs
    if (threadX >= GridNx - 1 || threadY >= GridNy - 1 || threadX == 0 || threadY == 0)
        return;

    // Compute the linear index assuming that X,Y then Z memory ordering
    const size_t idx = threadY * GridNx + threadX;

    long nodeIndex = idx;
    for (size_t z = 1; z < GridNz-1; z++)
    {        
        if (idx < size && (threadX + threadY + z) == s)
        {            
            long m0 = nodeIndex + z * GridXY;
            
            if (c0[m0] > 0)
            {
                //printf("236: threadX + threadY + z = %d \n", threadX + threadY + z);
                long m2 = m0 - 1;
                long m4 = m0 - GridNx;
                long m6 = m0 - GridXY;

                r[m0] = (omega * (c2[m0] * r[m2] + c4[m0] * r[m4] + c6[m0] * r[m6]) + r[m0]) / ((0.5 * omega + 1) * c0[m0]);                
                //printf("243: r[%d] = %lf\n", m0, r[m0]);
            }
        }        
    }
}


/// <summary>
/// ���, ������ �� Nx+Ny+Nz �� 0
/// </summary>
/// <param name="c"></param>
/// <param name="size">���-�� ��������� ������� s</param>
/// <param name="s">i+j+k</param>
/// <returns></returns>
__global__ void ptmKernel2(double* r, double* c0, double* c1, double* c3, double* c5, unsigned int size, int s, double omega)
{
    // Compute the offset in each dimension
    const size_t threadX = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t threadY = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Make sure that you are not actually outs
    if (threadX >= GridNx - 1 || threadY >= GridNy - 1 || threadX == 0 || threadY == 0)
        return;

    // Compute the linear index assuming that X,Y then Z memory ordering
    const size_t idx = threadY * GridNx + threadX;

    long nodeIndex = idx;
    for (size_t z = GridNz - 2; z >= 1; z--)
    {
        if (idx < size && (threadX + threadY + z) == s)
        {            
            long m0 = nodeIndex + z * GridXY;
            
            if (c0[m0] > 0)
            {                
                long m1 = m0 + 1;                
                long m3 = m0 + GridNx;                
                long m5 = m0 + GridXY;

                r[m0] = (omega * (c1[m0] * r[m1] + c3[m0] * r[m3] + c5[m0] * r[m5]) + r[m0] * c0[m0]) / ((0.5 * omega + 1) * c0[m0]);                
            }
        }
    }
}


__global__ void awrRrKernel(double* Awr, double* Rr, double* crr, double* r, double* c0, double* c1, double* c2, double* c3, double* c4, double* c5, double* c6, unsigned int size)
{
    // Compute the offset in each dimension
    const size_t threadX = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t threadY = blockDim.y * blockIdx.y + threadIdx.y;

    // Make sure that you are not actually outs
    if (threadX >= GridNx - 1 || threadY >= GridNy - 1 || threadX == 0 || threadY == 0)
        return;

    // Compute the linear index assuming that X,Y then Z memory ordering
    const size_t idx = threadY * GridNx + threadX;

    long nodeIndex = idx;
    for (size_t z = 1; z < GridNz - 1; z++)
    {
        if (idx < size)
        {
            long m0 = nodeIndex + z * GridXY;

            if (c0[m0] > 0)
            {
                long m1 = m0 + 1;
                long m2 = m0 - 1;
                long m3 = m0 + GridNx;
                long m4 = m0 - GridNx;
                long m5 = m0 + GridXY;
                long m6 = m0 - GridXY;

                Awr[m0] = (c0[m0] * r[m0] - ( c1[m0] * r[m1] + c2[m0] * r[m2] + c3[m0] * r[m3] + c4[m0] * r[m4] + c5[m0] * r[m5] + c6[m0] * r[m6])) * r[m0];
                double rr = 0.5 * c0[m0] * r[m0] - (c1[m0] * r[m1] + c3[m0] * r[m3] + c5[m0] * r[m5]);
                Rr[m0]  = rr * rr / c0[m0];
                crr[m0] = c0[m0] * r[m0] * r[m0];
            }
        }
    }

    /*__syncthreads();
    if (threadX == 1 && threadY == 1)
    {
        printf("\n\n ------------------- Print3dArrayDouble(Awr) ---------------------\n\n");
        Print3dArrayDouble(Awr);
    }

    __syncthreads();
    if (threadX == 1 && threadY == 1)
    {
        printf("\n\n ------------------- Print3dArrayDouble(Rr) ---------------------\n\n");
        Print3dArrayDouble(Rr);
    }

    __syncthreads();
    if (threadX == 1 && threadY == 1)
    {
        printf("\n\n ------------------- Print3dArrayDouble(crr) ---------------------\n\n");
        Print3dArrayDouble(crr);
    }*/
}

void Print3dArray(int* host_c)
{
    for (size_t k = 0; k < GridNz; k++)
    {
        printf("\n--------------------------------------\n");
        printf("------------ k = %d ------------------\n", k);
        printf("--------------------------------------\n");
        for (size_t j = 0; j < GridNy; j++)
        {
            printf("------------ j = %d ------------------\n", j);
            for (size_t i = 0; i < GridNx; i++)
            {
                printf("%d\t", host_c[i + j * GridNx + k * GridXY]);
            }
            printf("\n");
        }
    }
}

__host__ __device__ void Print3dArrayDouble(double* host_c)
{
    for (size_t k = 0; k < GridNz; k++)
    {
        printf("\n--------------------------------------\n");
        printf("------------ k = %d ------------------\n", k);
        printf("--------------------------------------\n");
        for (size_t j = 0; j < GridNy; j++)
        {
            printf("------------ j = %d ------------------\n", j);
            for (size_t i = 0; i < GridNx; i++)
            {
                printf("%lf\t", host_c[i + j * GridNx + k * GridXY]);
            }
            printf("\n");
        }
    }
}

void PtmTest()
{
    printf("------------------���� ������������ ����������----------------\n");
    // 1. ��������� ������ ������� ������
    int size = GridN; // ���-�� ���������
    size_t sizeInBytesInt = size * sizeof(int);   // ������ ������� int � ������
    size_t sizeInBytesDouble = size * sizeof(double);// ������ ������� double � ������

    // 2. �������� ������ ��� ������ � ���    
    double* host_c0 = (double*)malloc(sizeInBytesDouble);
    double* host_c1 = (double*)malloc(sizeInBytesDouble);
    double* host_c2 = (double*)malloc(sizeInBytesDouble);
    double* host_c3 = (double*)malloc(sizeInBytesDouble);
    double* host_c4 = (double*)malloc(sizeInBytesDouble);
    double* host_c5 = (double*)malloc(sizeInBytesDouble);
    double* host_c6 = (double*)malloc(sizeInBytesDouble);
    double* host_u = (double*)malloc(sizeInBytesDouble);
    double* host_f = (double*)malloc(sizeInBytesDouble);
    double* host_r = (double*)malloc(sizeInBytesDouble);
    double* host_Awr = (double*)malloc(sizeInBytesDouble);
    double* host_Rr  = (double*)malloc(sizeInBytesDouble);
    double* host_crr = (double*)malloc(sizeInBytesDouble);
    int* host_s = (int*)malloc(sizeInBytesInt);

    // 2a ������������� ��������
    for (size_t k = 0; k < GridNz; k++)
    {
        for (size_t j = 0; j < GridNy; j++)
        {
            for (size_t i = 0; i < GridNx; i++)
            {
                int m0 = i + j * GridNx + k * GridXY;
                host_c0[m0] = 4;
                host_c1[m0] = -1;
                host_c2[m0] = -1;
                host_c3[m0] = -1;
                host_c4[m0] = -1;
                host_c5[m0] = -1;
                host_c6[m0] = -1;
                host_u[m0] = 0;
                host_f[m0] = 10;
                host_r[m0] = 0;
                host_Awr[m0] = 0;
                host_Rr[m0] = 0;
                host_crr[m0] = 0;
                host_s[m0] = 1;
            }
        }
    }


    // 3. �������� ������ ��� ������� �� ����������    
    double* dev_c0 = NULL;
    hipMalloc((void**)&dev_c0, sizeInBytesDouble);

    double* dev_c1 = NULL;
    hipMalloc((void**)&dev_c1, sizeInBytesDouble);

    double* dev_c2 = NULL;
    hipMalloc((void**)&dev_c2, sizeInBytesDouble);

    double* dev_c3 = NULL;
    hipMalloc((void**)&dev_c3, sizeInBytesDouble);

    double* dev_c4 = NULL;
    hipMalloc((void**)&dev_c4, sizeInBytesDouble);

    double* dev_c5 = NULL;
    hipMalloc((void**)&dev_c5, sizeInBytesDouble);

    double* dev_c6 = NULL;
    hipMalloc((void**)&dev_c6, sizeInBytesDouble);

    double* dev_u = NULL;
    hipMalloc((void**)&dev_u, sizeInBytesDouble);

    double* dev_f = NULL;
    hipMalloc((void**)&dev_f, sizeInBytesDouble);

    double* dev_r = NULL;
    hipMalloc((void**)&dev_r, sizeInBytesDouble);

    double* dev_Awr = NULL;
    hipMalloc((void**)&dev_Awr, sizeInBytesDouble);

    double* dev_Rr = NULL;
    hipMalloc((void**)&dev_Rr, sizeInBytesDouble);

    double* dev_crr = NULL;
    hipMalloc((void**)&dev_crr, sizeInBytesDouble);

    int* dev_s = NULL;
    hipMalloc((void**)&dev_s, sizeInBytesInt);    
    
    // 4. �������� ������� �� ��� � GPU
    hipMemcpy(dev_c0, host_c0, sizeInBytesDouble, hipMemcpyHostToDevice);
    hipMemcpy(dev_c1, host_c1, sizeInBytesDouble, hipMemcpyHostToDevice);
    hipMemcpy(dev_c2, host_c2, sizeInBytesDouble, hipMemcpyHostToDevice);
    hipMemcpy(dev_c3, host_c3, sizeInBytesDouble, hipMemcpyHostToDevice);
    hipMemcpy(dev_c4, host_c4, sizeInBytesDouble, hipMemcpyHostToDevice);
    hipMemcpy(dev_c5, host_c5, sizeInBytesDouble, hipMemcpyHostToDevice);
    hipMemcpy(dev_c6, host_c6, sizeInBytesDouble, hipMemcpyHostToDevice);
    hipMemcpy(dev_u, host_u, sizeInBytesDouble, hipMemcpyHostToDevice);
    hipMemcpy(dev_f, host_f, sizeInBytesDouble, hipMemcpyHostToDevice);
    hipMemcpy(dev_r, host_r, sizeInBytesDouble, hipMemcpyHostToDevice);
    hipMemcpy(dev_Awr, host_Awr, sizeInBytesDouble, hipMemcpyHostToDevice);
    hipMemcpy(dev_Rr,  host_Rr,  sizeInBytesDouble, hipMemcpyHostToDevice);
    hipMemcpy(dev_crr, host_crr, sizeInBytesDouble, hipMemcpyHostToDevice);
    hipMemcpy(dev_s, host_s, sizeInBytesInt, hipMemcpyHostToDevice);
    

    // 5. ����������� ��������� ������ CUDA
    dim3 blocks(GridNx, GridNy);
    
    int it = 1;
    int host_isGreater = 0;
    int* dev_isGreater = NULL;
    hipMalloc((void**)&dev_isGreater, sizeof(int));

    float gpuTime = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    do
    {
        printf("-----------------------------------------------------------\n");
        printf("-------------Start of PTM Iteration------------------------\n");
        
        hipEventRecord(start, 0);

        // ���������� ������� �������
        nevyazkaKernel << < blocks, 1 >> > (dev_r, dev_c0, dev_c1, dev_c2, dev_c3, dev_c4, dev_c5, dev_c6, dev_f, dev_u, GridN);
        hipDeviceSynchronize();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpuTime, start, stop);
        printf("nevyazkaKernelTime = %f\n", gpuTime);


        hipEventRecord(start, 0);

        // �����������, ��������� �� ���� �� ���� ������� ������� ������� ������������ �������� ������        
        nevyazkaGreaterEpsKernel << <blocks, 1 >> > (dev_isGreater, dev_r, GridN, EPS);
        hipDeviceSynchronize();
        hipMemcpy(&host_isGreater, dev_isGreater, sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        //printf("host_isGreater = %d----------\n", host_isGreater);
        // ... while()

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpuTime, start, stop);
        printf("nevyazkaGreaterEpsKernel = %f\n", gpuTime);


        //hipEventRecord(start, 0);

        // 6. ����� ���������    
        double omega = 0.05;// �������� ��������� ��������????????????????????????
        double tay = 2 * omega;

        printf("--- ptmKernel1 Starting... ---\n", gpuTime);
        for (size_t i = 3; i < GridNx + GridNy + GridNz - 3; i++)
        {
            hipEventRecord(start, 0);
            ptmKernel1 << < blocks, 1 >> > (dev_r, dev_c0, dev_c2, dev_c4, dev_c6, GridN, i, omega);
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&gpuTime, start, stop);
            printf("ptmKernel1. i = %d; gpuTime = %f\n", i, gpuTime);
        }
        printf("--- ptmKernel1 End ---\n", gpuTime);
        hipDeviceSynchronize();

        /*hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpuTime, start, stop);
        printf("ptmKernel1 = %f\n", gpuTime);*/


        hipEventRecord(start, 0);

        for (size_t i = GridNx + GridNy + GridNz - 3; i >= 3; i--)
        {
            ptmKernel2 << < blocks, 1 >> > (dev_r, dev_c0, dev_c1, dev_c3, dev_c5, GridN, i, omega);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpuTime, start, stop);
        printf("ptmKernel2 = %f\n", gpuTime);


        hipEventRecord(start, 0);

        // ��������� ��������� ������������
        printf("--- awrRrKernel Starting... ---\n");
        awrRrKernel << < blocks, 1 >> > (dev_Awr, dev_Rr, dev_crr, dev_r, dev_c0, dev_c1, dev_c2, dev_c3, dev_c4, dev_c5, dev_c6, GridN);
        hipDeviceSynchronize();
        printf("--- awrRrKernel Ended ---\n");

        printf("--- RwRw = Reduce(dev_Rr, GridN); Starting... ---\n");
        double RwRw = Reduce(dev_Rr, GridN);
        printf("--- RwRw = Reduce(dev_Rr, GridN); Ended ---\n");
        printf("--- Aww = Reduce(dev_Awr, GridN); Starting... ---\n");
        double Aww = Reduce(dev_Awr, GridN);
        printf("--- Aww = Reduce(dev_Awr, GridN); Ended ---\n");
        printf("--- ww = Reduce(dev_crr, GridN); Starting... ---\n");
        double ww = Reduce(dev_crr, GridN);
        printf("--- ww = Reduce(dev_crr, GridN); Ended ---\n");
        //printf("RwRw = %lf\n", RwRw);
        //printf("Aww = %lf\n", Aww);
        //printf("ww = %lf\n", ww);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpuTime, start, stop);
        printf("awrRrKernel + RwRw + Aww + ww = %f\n", gpuTime);


        hipEventRecord(start, 0);
        if (ww > 0)
        {
            tay = 2 * omega + ww / Aww;
            omega = sqrt(ww / RwRw);
        }

        // ���������� dev_u
        uKernel << < blocks, 1 >> > (dev_u, dev_r, GridN, tay);                       

        it++;

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpuTime, start, stop);
        printf("uKernel = %f\n", gpuTime);
        printf("-------------End of PTM Iteration------------------------\n");
    } while (host_isGreater > 0 && it < 2/*200*/);

    // �������� ������ � ������������ ���������� �� ������ GPU � ���
    //hipMemcpy(host_r, dev_r, sizeInBytesDouble, hipMemcpyDeviceToHost);
    hipMemcpy(host_u, dev_u, sizeInBytesDouble, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    // ������� �� ������� ������ � ������������ ����������
    //Print3dArrayDouble(host_r);
    //Print3dArrayDouble(host_u);

    // ������� ������ ������
    free(host_c0);
    free(host_c1);
    free(host_c2);
    free(host_c3);
    free(host_c4);
    free(host_c5);
    free(host_c6);
    free(host_u);
    free(host_f);
    free(host_r);
    free(host_Awr);
    free(host_Rr);
    free(host_crr);
    free(host_s);

    hipFree(dev_c0);
    hipFree(dev_c1);
    hipFree(dev_c2);
    hipFree(dev_c3);
    hipFree(dev_c4);
    hipFree(dev_c5);
    hipFree(dev_c6);
    hipFree(dev_u);
    hipFree(dev_f);
    hipFree(dev_r);
    hipFree(dev_Awr);
    hipFree(dev_Rr);
    hipFree(dev_crr);
    hipFree(dev_s);

    // ���������� ���������� CUDA
    hipDeviceReset();    

    printf("--------------���� ������������ ���������� (�����)------------\n");
}

void ReductionTest()
{
    // 1. ��������� ������ ������� ������
    int size = 100000; // ���-�� ���������  
    
    size_t sizeInBytesDouble = size * sizeof(double);// ������ ������� double � ������

    // 2. �������� ������ ��� ������ � ���    
    double* host_a = (double*)malloc(sizeInBytesDouble);
 
    // 2a ������������� �������, ���������� ����� ��������� �������
    double host_a_sum = 0;
    for (size_t k = 0; k < size; k++)
    {
        host_a[k] = k + 0.2;            
        host_a_sum += host_a[k];
    }

    // 3. �������� ������ ��� ������ �� ����������    
    double* dev_a = NULL;
    hipMalloc((void**)&dev_a, sizeInBytesDouble);

    // 4. �������� ������ �� ��� � GPU
    hipMemcpy(dev_a, host_a, sizeInBytesDouble, hipMemcpyHostToDevice);

    double dev_a_sum = Reduce(dev_a, size);

    printf("host_a_sum = %lf\ndev_a_sum = %lf\n", host_a_sum, dev_a_sum);
}
