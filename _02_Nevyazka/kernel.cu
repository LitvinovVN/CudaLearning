#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <ctime>

#define BLOCK_SIZE 1000

__global__ 
void nevyazkaKernel(double* r, double* rMax, double* f, double* u)
{
    int tid = threadIdx.x;

    r[tid] = f[tid] - u[tid];

    //printf("treadId = %d: r[i] = %lf - %lf = %lf\n",tid, f[tid], u[tid], r[tid]);
    //printf("\n%lf; ", rMax[0]);
    
    __syncthreads();
      


    __shared__ double data[BLOCK_SIZE];    
    data[tid] = r[tid];
    //printf("\ndata[%d]=%lf; ",tid, data[tid]);
    
    //printf("29: treadId = %d: r[tid] = %lf; data[tid] = %lf\n", tid, r[tid], data[tid]);
    
    __syncthreads();

    for (int s = blockDim.x / 2; s > 32; s>>=1 )
    {
        if (tid < s)
        {
            if (fabs(data[tid]) < fabs(data[tid + s]))
            {
                //printf("38: treadId = %d: r[tid] = %lf; data[tid] = %lf --- data[tid + s] = %lf \n", tid, r[tid], data[tid], data[tid + s]);
                data[tid] = fabs(data[tid + s]);                
            }
        }            

        __syncthreads();               
    }
    
    //printf("42: treadId = %d: r[tid] = %lf; data[tid] = %lf\n", tid, r[tid], data[tid]);

    if (tid < 32)
    {
        if (fabs(data[tid]) < fabs(data[tid + 32]))
            data[tid] = fabs(data[tid + 32]);

        if (fabs(data[tid]) < fabs(data[tid + 16]))
            data[tid] = fabs(data[tid + 16]);

        if (fabs(data[tid]) < fabs(data[tid + 8]))
            data[tid] = fabs(data[tid + 8]);

        if (fabs(data[tid]) < fabs(data[tid + 4]))
            data[tid] = fabs(data[tid + 4]);

        if (fabs(data[tid]) < fabs(data[tid + 2]))
            data[tid] = fabs(data[tid + 2]);

        if (fabs(data[tid]) < fabs(data[tid + 1]))
            data[tid] = fabs(data[tid + 1]);
    }

    if (tid == 0)
    {
        rMax[0] = data[0];
        printf("\n\nrMax[0] = %lf;\n\n", rMax[0]);
    }//*/
}




void init_f(double* f, int array_size)
{
    for (size_t i = 0; i < array_size; i++)
    {
        f[i] = 1000 + i;
        //printf("%.1lf; ", f[i]);
    }
}

void init_u(double* u, double* f, int array_size)
{
    for (size_t i = 0; i < array_size; i++)
    {
        // ���������� ��������� �������������� ����� �� -100 �� 100
        double delta = ((double)(rand()) / RAND_MAX * 200 - 100);

        u[i] = f[i] + delta;
        //printf("%.1lf; ", r[i]);
    }
}

void init_r(double* r, int array_size)
{
    for (size_t i = 0; i < array_size; i++)
    {
        r[i] = 0;
        //printf("%.1lf; ", r[i]);
    }
}

/// <summary>
/// ��������� ������ ������� � ��� ������������ ��������
/// </summary>
/// <param name="r">������ ������� (�������)</param>
/// <param name="rMax">������������ �������� ������� ������� (�������)</param>
/// <param name="f">������ �������� �������</param>
/// <param name="u">������ ��������������-������������ �������� �������</param>
/// <param name="array_size">������ ��������</param>
/// <returns></returns>
hipError_t calcNevyazkaWithCuda(double* r, double &rMax, double* f, double* u, int array_size)
{
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);



    double* dev_r = 0;
    double* dev_f = 0;
    double* dev_u = 0;
    double* dev_rMax = 0;
    hipError_t cudaStatus;

    // ����� GPU ��� �������
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    ///////////////////////////////////////////////////////////////

    // ��������� ������� ������ �� GPU (2 �����, 1 �����)
    size_t array_size_in_bytes = array_size * sizeof(double);

    cudaStatus = hipMalloc((void**)&dev_r, array_size_in_bytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_f, array_size_in_bytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_u, array_size_in_bytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_rMax, sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    //////////////////////////////////////////////////////////

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_f, f, array_size_in_bytes, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_u, u, array_size_in_bytes, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    /////////////////////////////////////////////////////////////    

    

    // Launch a kernel on the GPU with one thread for each element.
    nevyazkaKernel << <1, array_size >> > (dev_r, dev_rMax, dev_f, dev_u);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    printf("Time for the kernel: %f ms\n", time);
    
    printf("\n---------------------\n");
    printf("����� ���������� nevyazkaKernel: %f ms\n", time);
    printf("---------------------\n");

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    //////////////////////////////////////////////////////////////

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(r, dev_r, array_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    double* rMaxArray = new double[1];
    cudaStatus = hipMemcpy(rMaxArray, dev_rMax, sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    

    /*for (int i = 0; i < array_size; i++)
    {
        printf("r[%d] = %lf - %lf = %lf\n", i, f[i], u[i], r[i]);
    }*/


Error:
    hipFree(dev_r);
    hipFree(dev_f);
    hipFree(dev_u);

    return cudaStatus;
}


int main()
{
    // ������ �������
    int array_size = BLOCK_SIZE;

    // ������ ��������� �������� �������
    double* f = new double[array_size];
    // ������ ������������ �������� �������
    double* u = new double[array_size];
    // ������ �������
    double* r = new double[array_size];
    // ������������ �������� �������
    double rMax = 0;

    // ������������� ������� f ���������� �� 1000 �� (1000 + array_size - 1)
    init_f(f, array_size);

    // ������������� ������� u c��������������� ���������� ������� f �� ��������� �����������
    init_u(u, f, array_size);

    // ������������� ������� r �������� ����������
    init_r(r, array_size);

    // ���������� ������� ������� � ��� ������������� �������� �� Cuda
    hipError_t cudaStatus = calcNevyazkaWithCuda(r, rMax, f, u, array_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "calcNevyazkaWithCuda failed!");
        return 1;
    }        
}